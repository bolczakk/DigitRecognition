#include "hip/hip_runtime.h"
#include "Layer.cuh"

Layer::Layer(int inputCount, int outputCount) {
    this->inputs.resize(inputCount);
    this->outputs.resize(outputCount);
    this->weights.resize(inputCount);
    this->biases.resize(outputCount);
    initWeightsBiases(inputCount, outputCount);
}

__device__ float sigmoid(float x) {
    return 1.0f / (1.0f + exp(-x));
}

// CUDA kernel dla oblicze� wyj�ciowych
__global__ void computeOutputs(
    float* d_inputs, float* d_weights, float* d_biases, float* d_outputs,
    int inputSize, int outputSize) {
    int outputIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (outputIdx < outputSize) {
        float sum = d_biases[outputIdx];
        for (int i = 0; i < inputSize; i++) {
            sum += d_inputs[i] * d_weights[i * outputSize + outputIdx];
        }
        d_outputs[outputIdx] = sigmoid(sum);
    }
}

std::vector<float> Layer::feedForward(std::vector<float> givenInputs, Layer& layer) {
    int inputSize = layer.inputs.size();
    int outputSize = layer.outputs.size();
    float* d_inputs;
    float* d_weights;
    float* d_biases;
    float* d_outputs;

    for (int i = 0; i < layer.inputs.size(); i++) {
        layer.inputs[i] = givenInputs[i];
    }

    // Rozmiar wag w postaci 1D (inputSize * outputSize)
    size_t weightsSize = inputSize * outputSize * sizeof(float);
    size_t inputsSize = inputSize * sizeof(float);
    size_t biasesSize = outputSize * sizeof(float);
    size_t outputsSize = outputSize * sizeof(float);

    hipMalloc(&d_inputs, inputsSize);
    hipMalloc(&d_weights, weightsSize);
    hipMalloc(&d_biases, biasesSize);
    hipMalloc(&d_outputs, outputsSize);

    // Przekszta�cenie wag do 1D
    std::vector<float> flatWeights(inputSize * outputSize);
    for (int i = 0; i < inputSize; i++) {
        for (int j = 0; j < outputSize; j++) {
            flatWeights[i * outputSize + j] = layer.weights[i][j];
        }
    }

    // Kopiowanie danych z hosta (CPU) do urz�dzenia (GPU)
    hipMemcpy(d_inputs, layer.inputs.data(), inputsSize, hipMemcpyHostToDevice);
    hipMemcpy(d_weights, flatWeights.data(), weightsSize, hipMemcpyHostToDevice);
    hipMemcpy(d_biases, layer.biases.data(), biasesSize, hipMemcpyHostToDevice);

    // Obliczenia na GPU: konfiguracja siatki i blok�w
    int threadsPerBlock = 256;
    int blocksPerGrid = (outputSize + threadsPerBlock - 1) / threadsPerBlock;

    computeOutputs << <blocksPerGrid, threadsPerBlock >> > (
        d_inputs, d_weights, d_biases, d_outputs,
        inputSize, outputSize);

    // Pobranie wynik�w z GPU
    std::vector<float> outputs(outputSize);
    hipMemcpy(outputs.data(), d_outputs, outputsSize, hipMemcpyDeviceToHost);

    // Zwolnienie pami�ci GPU
    hipFree(d_inputs);
    hipFree(d_weights);
    hipFree(d_biases);
    hipFree(d_outputs);

    return outputs;
}

void Layer::initWeightsBiases(int inputCount, int outputCount) {
    for (int i = 0; i < inputCount; i++) {
        weights[i].resize(outputCount);
        for (int j = 0; j < outputCount; j++) {
            weights[i][j] = randomFloat();
        }
    }
    for (int i = 0; i < biases.size(); i++) {
        biases[i] = randomFloat();
    }
}

float Layer::randomFloat() {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0f, 1.0f);
    float randomFloat = dis(gen);

    return randomFloat;
}

float Layer::sigmoid(float x) {
    return 1.0f / (1.0f + std::exp(-x));
}

float Layer::sigmoid_derivative(float x) {
    return x * (1 - x);
}

float Layer::RELU(float x) {
    return std::max(0.0f, x);
}

float Layer::RELU_derivative(float x) {
    return x > 0 ? 1.0f : 0.0f;
}

void Layer::saveLayer(std::ofstream& file) {
    for (const auto& bias : biases) {
        file.write(reinterpret_cast<const char*>(&bias), sizeof(bias));
    }
    for (const auto& weightRow : weights) {
        for (const auto& weight : weightRow) {
            file.write(reinterpret_cast<const char*>(&weight), sizeof(weight));
        }
    }
}

void Layer::loadLayer(std::ifstream& file) {
    for (auto& bias : biases) {
        file.read(reinterpret_cast<char*>(&bias), sizeof(bias));
    }
    for (auto& weightRow : weights) {
        for (auto& weight : weightRow) {
            file.read(reinterpret_cast<char*>(&weight), sizeof(weight));
        }
    }
}

std::vector<float> Layer::calculateError(const std::vector<float>& expectedOutputs) {
    std::vector<float> errors(expectedOutputs.size());
    for (size_t i = 0; i < outputs.size(); i++) {
        errors[i] = outputs[i] - expectedOutputs[i];
    }
    return errors;
}

void Layer::calculateOutputDeltas(const std::vector<float> error) {
    deltas.resize(outputs.size());
    for (int i = 0; i < deltas.size(); i++) {
        deltas[i] = 2 * error[i] * sigmoid_derivative(outputs[i]);
    }
}

__global__ void calculateHiddenDeltasKernel(
    float* deltas,
    const float* outputs,
    const float* nextDeltas,
    const float* nextWeights,
    int outputSize,
    int nextOutputSize) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < outputSize) {
        float sum = 0.0f;
        for (int j = 0; j < nextOutputSize; ++j) {
            sum += nextDeltas[j] * nextWeights[idx * nextOutputSize + j];
        }
        deltas[idx] = sum * outputs[idx] * (1 - outputs[idx]); // sigmoid_derivative
    }
}


void Layer::calculateHiddenDeltas(Layer& nextLayer) {
    int outputSize = outputs.size();
    int nextOutputSize = nextLayer.outputs.size();

    // Alokacja pami�ci na GPU
    float* d_deltas, * d_outputs, * d_nextDeltas, * d_nextWeights;

    hipMalloc(&d_deltas, outputSize * sizeof(float));
    hipMalloc(&d_outputs, outputSize * sizeof(float));
    hipMalloc(&d_nextDeltas, nextOutputSize * sizeof(float));
    hipMalloc(&d_nextWeights, outputSize * nextOutputSize * sizeof(float));

    // Kopiowanie danych na GPU
    hipMemcpy(d_outputs, outputs.data(), outputSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_nextDeltas, nextLayer.deltas.data(), nextOutputSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_nextWeights, nextLayer.weights[0].data(),
        outputSize * nextOutputSize * sizeof(float), hipMemcpyHostToDevice);

    // Wywo�anie kernela
    int threadsPerBlock = 256;
    int blocksPerGrid = (outputSize + threadsPerBlock - 1) / threadsPerBlock;
    calculateHiddenDeltasKernel << <blocksPerGrid, threadsPerBlock >> > (
        d_deltas, d_outputs, d_nextDeltas, d_nextWeights, outputSize, nextOutputSize);

    // Kopiowanie wynik�w z GPU do pami�ci hosta
    hipMemcpy(deltas.data(), d_deltas, outputSize * sizeof(float), hipMemcpyDeviceToHost);

    // Zwolnienie pami�ci GPU
    hipFree(d_deltas);
    hipFree(d_outputs);
    hipFree(d_nextDeltas);
    hipFree(d_nextWeights);
}

float Layer::countSum(Layer& nextLayer, int i) {
    float sum = 0.0f;
    for (size_t j = 0; j < nextLayer.outputs.size(); ++j) {
        sum += nextLayer.deltas[j] * nextLayer.weights[i][j];
    }
    return sum;
}

__global__ void updateBiases(float* d_biases, float* d_deltas, float learningRate, int outputSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < outputSize) {
        d_biases[idx] -= d_deltas[idx] * learningRate;
    }
}

// CUDA kernel do aktualizacji wag
__global__ void updateWeights(
    float* d_weights, float* d_inputs, float* d_deltas,
    float learningRate, int inputSize, int outputSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int row = idx / outputSize;  // Indeks wiersza (wej�cie)
    int col = idx % outputSize;  // Indeks kolumny (wyj�cie)

    if (row < inputSize && col < outputSize) {
        int weightIdx = row * outputSize + col;
        d_weights[weightIdx] -= d_deltas[col] * d_inputs[row] * learningRate;
    }
}

void Layer::updateWeightsAndBiases(float learningRate) {
    int inputSize = inputs.size();
    int outputSize = biases.size();

    // Alokacja pami�ci na GPU
    float* d_biases;
    float* d_deltas;
    float* d_weights;
    float* d_inputs;

    size_t biasesSize = outputSize * sizeof(float);
    size_t deltasSize = outputSize * sizeof(float);
    size_t weightsSize = inputSize * outputSize * sizeof(float);
    size_t inputsSize = inputSize * sizeof(float);

    hipMalloc(&d_biases, biasesSize);
    hipMalloc(&d_deltas, deltasSize);
    hipMalloc(&d_weights, weightsSize);
    hipMalloc(&d_inputs, inputsSize);

    // Przekszta�cenie wag do 1D
    std::vector<float> flatWeights(inputSize * outputSize);
    for (int i = 0; i < inputSize; i++) {
        for (int j = 0; j < outputSize; j++) {
            flatWeights[i * outputSize + j] = weights[i][j];
        }
    }

    // Kopiowanie danych z CPU na GPU
    hipMemcpy(d_biases, biases.data(), biasesSize, hipMemcpyHostToDevice);
    hipMemcpy(d_deltas, deltas.data(), deltasSize, hipMemcpyHostToDevice);
    hipMemcpy(d_weights, flatWeights.data(), weightsSize, hipMemcpyHostToDevice);
    hipMemcpy(d_inputs, inputs.data(), inputsSize, hipMemcpyHostToDevice);

    // Konfiguracja siatki i blok�w
    int threadsPerBlock = 256;
    int biasBlocks = (outputSize + threadsPerBlock - 1) / threadsPerBlock;
    int weightBlocks = ((inputSize * outputSize) + threadsPerBlock - 1) / threadsPerBlock;

    // Uruchomienie kernel�w
    updateBiases << <biasBlocks, threadsPerBlock >> > (d_biases, d_deltas, learningRate, outputSize);
    updateWeights << <weightBlocks, threadsPerBlock >> > (
        d_weights, d_inputs, d_deltas, learningRate, inputSize, outputSize);

    // Pobranie zaktualizowanych danych z GPU na CPU
    hipMemcpy(biases.data(), d_biases, biasesSize, hipMemcpyDeviceToHost);

    hipMemcpy(flatWeights.data(), d_weights, weightsSize, hipMemcpyDeviceToHost);

    // Przekszta�cenie wag z 1D na 2D
    for (int i = 0; i < inputSize; i++) {
        for (int j = 0; j < outputSize; j++) {
            weights[i][j] = flatWeights[i * outputSize + j];
        }
    }

    // Zwolnienie pami�ci na GPU
    hipFree(d_biases);
    hipFree(d_deltas);
    hipFree(d_weights);
    hipFree(d_inputs);
}
